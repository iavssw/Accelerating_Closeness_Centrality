#include "hip/hip_runtime.h"
#define inf (999999999.0)
#include <stdio.h>
#include <math.h>
#include <stdlib.h>

#include <hip/hip_vector_types.h>

__global__ void firstpass(int n, int k, float* x) //, int* qx) 
{
    __shared__ float dBlck[1024]; //, qBlck[1024];
    
    float tmp = 0.00;
    int i = (threadIdx.x >> 5), j = threadIdx.x & 31;
    int ind1 = ((k << 5) + i) * n + (k << 5) + j; //, k1 = k << 5;

    dBlck[threadIdx.x] = x[ind1];
    //qBlck[threadIdx.x] = qx[ind1];

    for (int l = 0; l < 32; ++l) 
    {
        __syncthreads();
        tmp = dBlck[(i << 5) + l] + dBlck[(l << 5) + j];
        if (dBlck[threadIdx.x] > tmp) 
        {
            dBlck[threadIdx.x] = tmp;
            //qBlck[threadIdx.x] = l + k1;
        }
    }
    x[ind1] = dBlck[threadIdx.x];
    //qx[ind1] = qBlck[threadIdx.x];
}

__global__ void secondpass(int n, int k, float* x) //, int* qx) 
{
    __shared__ float dBlck[1024], cBlock[1024]; //qcBlck[1024]

    int i = (threadIdx.x >> 5), j = threadIdx.x & 31, k1 = (k << 5), skip = 0;

    dBlck[threadIdx.x] = x[(k1 + i) * n + k1 + j];
    float tmp = 0.00;

    if (blockIdx.x >= k) // jump over block computed in first pass
    { 
        skip = 1;
    }
    if (blockIdx.y == 0) 
    {
        int ind1 = (k1 + i) * n + ((blockIdx.x + skip) << 5) + j;
        cBlock[threadIdx.x] = x[ind1];
        //qcBlck[threadIdx.x] = qx[ind1];

        for (int l = 0; l < 32; ++l) 
        {
            __syncthreads();
            tmp = dBlck[(i << 5) + l] + cBlock[(l << 5) + j];
            if (cBlock[threadIdx.x] > tmp) 
            {
                cBlock[threadIdx.x] = tmp;
                //qcBlck[threadIdx.x] = l + k1;
            }
        }
        x[ind1] = cBlock[threadIdx.x];
        //qx[ind1] = qcBlck[threadIdx.x];

    } 
    else 
    {
        int ind1 = (((blockIdx.x + skip)<<5) + i) * n + k1 + j;
        cBlock[threadIdx.x] = x[ind1];
        //qcBlck[threadIdx.x] = qx[ind1];

        for (int l = 0; l < 32; ++l) 
        {
            __syncthreads();
            tmp = cBlock[(i << 5) + l] + dBlck[(l << 5) + j];

            if (cBlock[threadIdx.x] > tmp) 
            {
                cBlock[threadIdx.x] = tmp;
                //qcBlck[threadIdx.x] = l + k1;
            }
        }
        x[ind1] = cBlock[threadIdx.x];
        //qx[ind1] = qcBlck[threadIdx.x];
    }
}

__global__ void thirdpass(int n, int k, float* x)
{
    int i = (threadIdx.x >> 5), j = threadIdx.x & 31, skipx = 0, skipy = 0;

    __shared__ float dyBlck[1024], dxBlck[1024];

    float cBlock_local = 0.00f;

    if (blockIdx.x >= k) 
    {
        skipx = 1;
    }
    if (blockIdx.y >= k) 
    {
        skipy = 1;
    }

    dxBlck[threadIdx.x] = x[((k << 5) + i) * n + ((blockIdx.y + skipy) << 5) + j];
    dyBlck[threadIdx.x] = x[(((blockIdx.x + skipx) << 5) + i) * n + (k << 5) + j];

    __syncthreads();

    int ind1 = (((blockIdx.x + skipx) << 5) + i) * n + ((blockIdx.y + skipy) << 5) + j;

    cBlock_local = x[ind1];

    float tmp = 0.00f;
    for (int l = 0; l < 32; ++l) 
    {
        tmp = dyBlck[(i << 5) + l] + dxBlck[(l << 5) + j];
        cBlock_local = fminf(cBlock_local, tmp);
    }
    x[ind1] = cBlock_local;
}

// closeness centrality kernel:
//  this is just a matrix-vector multiplication with unit vector
//  plus a reciprocal on the resulting vector
// @n: number of vertices
// @x: shortest distance matrix:
//      x[i * n + j] is the shortest distance from vertex i to vertex j
// @cc: output array of closeness centrality for each vertex
__global__ void closeness_centrality(int n, float* x, float* cc)
{
    // each thread works on one vertex
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;  // assert(tid < n)

    float sum = 0.0f;
    if (tid < n) {
        for (int i = 0; i < n; i++) {
            if (tid != i) {
                float dis = x[tid * n + i];
                sum += 1 / dis; // use harmonic sum in case the graph is not strongly connected
            }
        }
        cc[tid] = sum;
    }
}

int main(int argc, char **argv) 
{

    hipEvent_t start, stop, fw_finished;
    float *host_A, *host_D, *dev_x, *A, *D, tolerance = 0.001, sum = 0, dt_ms = 0;
    int n = atoi(argv[1]), i, j, bk11 = 1, bk21 = n/32 - 1, bk22 = 2, bk31 = n/32 - 1, bk32 = n/32 - 1, k = 0;
    int *Q;
    float *host_cc, *dev_cc, *cc;
    double t1s, t2s, t3s, t4s, t5s;
    char runcpu = argv[2][0];

    printf("==========================================\n");
    printf("Running with %d nodes \n", n);
    printf("\n");

    hipMalloc(&dev_x, n * n * sizeof(float));
    hipMalloc(&dev_cc, n * sizeof(float));
    //hipMalloc(&dev_qx, n * n * sizeof(float));

    // Arrays for the CPU
    A = (float *) malloc(n * n * sizeof(float));    // weights (input graph)
    D = (float *) malloc(n * n * sizeof(float));    // shortest distances (results)
    Q = (int *) malloc(n * n * sizeof(int));        // does not affect computation result; used for debugging
    cc = (float*) malloc(n * sizeof(float));    // closeness centrality values

    // Arrays for the GPU
    host_A = (float *) malloc(n * n * sizeof(float));
    host_D = (float *) malloc(n * n * sizeof(float));
    host_cc = (float*) malloc(n * sizeof(float));

    // generate random graph
    srand(time(NULL));
    for (i = 0; i < n; ++i) 
    {
        for (j = 0; j < n; ++j) 
        {
            Q[i * n + j] = -1;
        }
        cc[i] = 0.0f;
    }
    for (i = 0; i < n; ++i) 
    {
        for (j = 0; j < n; ++j) 
        {
            if (i == j) 
            {
                A[i * n + j] = 0;
            } 
            else 
            {
                A[i * n + j] = 1200 * (float) rand() / RAND_MAX + 1;
                if (A[i * n + j] > 1000) 
                {
                    A[i * n + j] = inf;
                    Q[i * n + j] = -2;
                }
            }
            D[i * n + j] = A[i * n + j];
            host_A[i * n + j] = A[i * n + j];
        }
    }

    hipEventCreate(&start);
    hipEventCreate(&fw_finished);
    hipEventCreate(&stop);

    // First copy, CPU -> GPU

    hipEventRecord(start, 0);
    hipMemcpy(dev_x, host_A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(stop, 0);
    hipEventSynchronize(start); 
    hipEventSynchronize(stop);
    hipEventElapsedTime(&dt_ms, start, stop);
    printf("Transfer CPU -> GPU, time: %lf ms\n", dt_ms);
    sum+=dt_ms;
    t1s = dt_ms;

    // GPU calculation
    hipEventRecord(start, 0);
    dim3 bk2(n / 32 - 1, 2);
    dim3 bk3(n / 32 - 1, n / 32 - 1);
    int gputhreads = 1024;
    for (k = 0; k < n / 32; ++k) 
    {
        firstpass<<<1, gputhreads>>>(n, k, dev_x);
        secondpass<<<bk2, gputhreads>>>(n, k, dev_x);
        thirdpass<<<bk3, gputhreads>>>(n, k, dev_x);
    }
    hipEventRecord(fw_finished, 0);

    closeness_centrality<<<n / 256 + 1, 256>>>(n, dev_x, dev_cc);

    hipDeviceSynchronize(); // wait until all threads are done

    hipEventRecord(stop, 0);
    hipEventSynchronize(start); 
    hipEventSynchronize(fw_finished);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&dt_ms, start, stop);
    float fwtime, cctime;
    hipEventElapsedTime(&fwtime, start, fw_finished);
    hipEventElapsedTime(&cctime, fw_finished, stop);
    printf("Calculation time for GPU: %lf ms\n", dt_ms);
    printf("FW time: %lf ms; CC time: %lf ms\n", fwtime, cctime);
    sum+=dt_ms;
    t2s = dt_ms;

    // Second copy, GPU -> CPU
    
    hipEventRecord(start, 0);
    hipMemcpy(host_D, dev_x, n * n * sizeof (float), hipMemcpyDeviceToHost);
    hipMemcpy(host_cc, dev_cc, n * sizeof (float), hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(start); 
    hipEventSynchronize(stop);
    hipEventElapsedTime(&dt_ms, start, stop);
    printf("Transfer GPU -> CPU, time: %lf ms\n", dt_ms);
    sum+=dt_ms;
    t3s = dt_ms;

    printf("Total time: %lf ms\n\n----------------------------\n", sum);
    t4s = sum;
    
   // Running sequentially on CPU now

    if(runcpu == 'y')
    {
        printf("\n");
        printf("Sequential execution on CPU (could take a while)... \n");
        hipEventRecord(start, 0);
        for (k = 0; k < n; ++k) 
        {
            for (i = 0; i < n; ++i) 
            {
                for (j = 0; j < n; ++j) 
                {
                    if ((D[i * n + k] + D[k * n + j]) < D[i * n + j]) 
                    {
                        D[i * n + j] = D[i * n + k] + D[k * n + j];
                        Q[i * n + j] = k;
                    }
                }
            }
        }

        // compute closeness centrality
        for (i = 0; i < n; i++)
            for (j = 0; j < n; j++) {
                if (i != j) {
                    cc[i] += 1 / D[i * n + j];
                }
            }

        hipEventRecord(stop, 0);
        hipEventSynchronize(start); 
        hipEventSynchronize(stop);
        hipEventElapsedTime(&dt_ms, start, stop);
        printf("CPU time: %lf ms\n", dt_ms);
        t5s = dt_ms;
        printf("\n");

        // Result validation

        printf("FW: Comparing CPU results with GPU results...");
        for (i = 0; i < n; ++i) 
        {
            for (j = 0; j < n; ++j) 
            {
                if (abs(D[i * n + j] - host_D[i * n + j]) > tolerance) 
                {
                    printf("\nERROR: Different results in row i = %d and column j = %d, CPU result = %f, GPU result = %f \n", i, j, D[i * n + j], host_D[i * n + j]);
                }
            }
        }

        printf("CC: Comparing CPU results with GPU results...");
        for (i = 0; i < n; i++) {
            if (abs(cc[i] - host_cc[i]) > tolerance) {
                printf("\nERROR: Different results\n for vertex %d, CPU result = %f, GPU result = %f \n", i, cc[i], host_cc[i]);
            }
        }

        printf("Comparison complete! \n");
    }
    else
    {
        t5s = -1;
    }
    printf("Results are written to file resultsfwblocked.csv\n==========================================\n");
    FILE *fptr;
    fptr = fopen("resultsfwblocked.csv","a");

    fprintf(fptr,"%d,%d,%d,%d,%d,%d,%d,%lf,%lf,%lf,%lf,%lf,%lf,%lf\n",n, bk11, bk21, bk22, bk31, bk32, gputhreads, t1s, t2s, t3s, t4s, t5s, fwtime, cctime);
    fclose(fptr);
    return 0;
}

